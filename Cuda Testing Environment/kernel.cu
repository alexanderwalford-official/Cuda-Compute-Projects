﻿
#include "hip/hip_runtime.h"


#include <stdio.h>


int main()
{
    int a[] = { 1, 2 ,3 };
    int b[] = { 4, 5, 6 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    for (int i = 0; i < sizeof(c) / sizeof(int); i++) {
        c[i] = a[i] + b[i];
    }

    printf("Compiled on GPU.");

    return;
}