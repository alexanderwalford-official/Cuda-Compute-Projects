﻿// docs: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void vectAdd(int* a, int* b, int* c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void LinearSort(int* a, int* b) {
    int i = threadIdx.x;

    // TBA
}

void ComputeExampleA() {
    // arrys
    int a[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
    int b[] = { 4, 5, 6, 7, 8, 9, 10, 11, 12 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    // create GPU pointers
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    // alloc GPU mem
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // cpy vects into GPU
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);

    // grid size, block size
    vectAdd <<< 1, sizeof(a) / sizeof(int) >>>(cudaA, cudaB, cudaC);

    // cpy results from GPU, costly operation
    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    std::cout << "Compute Example A: DONE \n";
    return;
}

void ComputeExampleB() {
    // let's give the GPU a linear sorting algorithm

    // pointers
    int* cudaA;
    int* cudaB;

    // arrys
    int starting_arr[] = {1, 4, 2, 5, 7, 1, 3, 7};
    int sorted_arr[sizeof(starting_arr)];

    // alloc GPU mem
    hipMalloc(&cudaA, sizeof(starting_arr));
    hipMalloc(&cudaB, sizeof(sorted_arr));

    // cpy into GPU mem
    hipMemcpy(cudaA, starting_arr, sizeof(starting_arr), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, sorted_arr, sizeof(sorted_arr), hipMemcpyHostToDevice);

    // call sorting method on GPU
    LinearSort <<< 1, sizeof(starting_arr) / sizeof(int) >>>(cudaA, cudaB);

    // cpy results from GPU
    hipMemcpy(sorted_arr, cudaB, sizeof(sorted_arr), hipMemcpyDeviceToHost);

    std::cout << "Compute Example B: DONE \n";
    return;
}

int main()
{
    // disable / enable relevant methods here by commeting them in / out
    ComputeExampleA(); // addition example
    ComputeExampleB(); // linear sorting example
    return 0;
}
